// Template for Programming Assignment 2
// Use "module load cuda" to enable compilation with the Nvidia C compiler nvcc
// Use "nvcc -O3" to compile code; this can be done even on OSC login node (does not have a GPU)
// To execute compiled code, you must either use a batch submission to run on a node with GPU
// or obtain an interactive GPU-node by using: qsub -I -l walltime=0:59:00 -l nodes=1:gpus=1


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#define threshold 1e-8
#define n (4096)
// Change n to 4096 for final testing; 
//#define n (256)
// n is set to 256 since execution time of single thread template version is excessive
#define TILE_WIDTH 32

void init(void);
void ref(void);
void compare(int N, double *wref, double *w);
__global__ void test_kernel(int N, double *A, double *B, double *C);
double rtclock(void);

double a[n][n],b[n][n],c[n][n],cref[n][n];

int main(){

double clkbegin, clkend, t;
double *Ad,*Bd,*Cd;
int size;

  printf("Matrix Size = %d\n",n);

  init();
  
  clkbegin = rtclock();
  ref();
  clkend = rtclock();
  t = clkend-clkbegin;
  printf("Seq: Approx GFLOPS: %.1f ; Time = %.3f sec; cref[n/2][n/2-1] = %f; \n",
2.0*n*n*n/t/1e9,t,cref[n/2][n/2-1]);

  
  size = sizeof(double)*n*n;
  hipMalloc((void **) &Ad,size);
  hipMalloc((void **) &Bd,size);
  hipMalloc((void **) &Cd,size);
  hipMemcpy(Ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(Bd,b,size,hipMemcpyHostToDevice);
  clkbegin = rtclock();
  
  dim3 dimGrid(n/(2*TILE_WIDTH), n/(2*TILE_WIDTH));
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  
  test_kernel<<<dimGrid,dimBlock>>>(n,Ad,Bd,Cd);
  if (hipDeviceSynchronize() != hipSuccess) 
    printf ("Error return for test_kernel: Was execution done on a node with a GPU?\n");
  else
  {
   clkend = rtclock();
   t = clkend-clkbegin;
   hipMemcpy(c,Cd,size,hipMemcpyDeviceToHost);
   hipFree(Ad); hipFree(Bd); hipFree(Cd);
    printf("GPU: Approx GFLOPS: %.1f ; Time = %.3f sec; c[n/2][n/2-1] = %f; \n",
     2.0*n*n*n/t/1e9,t,c[n/2][n/2-1]);
    printf("Correctness Check for GPU solution:\n");
   compare(n, (double *) c,(double *) cref);
  }
}

__global__ void test_kernel(int N, double *A, double *B, double *C)
{
// using shared memory for storing TILES corresponding to this block in A & B. Each block is responsible for computing 4 TILES in C
__shared__ double As0[TILE_WIDTH][TILE_WIDTH];
__shared__ double As1[TILE_WIDTH][TILE_WIDTH];
__shared__ double Bs0[TILE_WIDTH][TILE_WIDTH+1];
__shared__ double Bs1[TILE_WIDTH][TILE_WIDTH+1];

// Block Index along x & y
int bx = blockIdx.x; int by = blockIdx.y;

//Thread Index along x & y
int tx = threadIdx.x; int ty = threadIdx.y;

// Rows & Columns in resultant matrix C computed by the threads in the block
int Row = by * 2 *TILE_WIDTH + ty;
int Column = bx * 2 * TILE_WIDTH + tx;

double Pvalue0 = 0;
double Pvalue1 = 0;
double Pvalue2 = 0;
double Pvalue3 = 0;

// breaking kernel into 'N/TILE_WIDTH' phases where 'm' is the current phase
for(int m=0; m<N/TILE_WIDTH; ++m) {
    
    // Bring elements from A & B to their corresponding shared memory
    As0[ty][tx] = A[Row*N+(m*TILE_WIDTH+tx)];
    As1[ty][tx] = A[(Row+TILE_WIDTH)*N+(m*TILE_WIDTH+tx)];
    
    Bs0[ty][tx] = B[((bx*2*TILE_WIDTH+ty)*N)+(m*TILE_WIDTH+tx)];
    Bs1[ty][tx] = B[((bx*2*TILE_WIDTH+TILE_WIDTH+ty)*N)+(m*TILE_WIDTH+tx)];
    
    //wait for the entire tiles in A & B to be in shared memory
    __syncthreads();
    
    // Accumulate subset of dot products
    for (int k=0; k<TILE_WIDTH; ++k) {
        
        Pvalue0 +=  As0[ty][k]*Bs0[tx][k];
        Pvalue1 +=  As0[ty][k]*Bs1[tx][k];
        Pvalue2 +=  As1[ty][k]*Bs0[tx][k];
        Pvalue3 +=  As1[ty][k]*Bs1[tx][k];
       
    }
    // wait for the entire values corresponding to this phase to be computed
        __syncthreads();

    }
     // write final values in C computed by this block to global memory
    C[Row*N+Column] = Pvalue0;
    C[Row*N+Column+TILE_WIDTH] = Pvalue1;
    C[(Row+TILE_WIDTH)*N+Column] = Pvalue2;
    C[(Row+TILE_WIDTH)*N+Column+TILE_WIDTH] = Pvalue3;
}

void ref(void)
{
int i,j,k;

  for (i=0;i<n;i++)
   for (j=0;j<n;j++)
    for(k=0;k<n;k++)
      cref[i][j] += a[i][k]*b[j][k];
}

void init(void)
{
int i,j;
for(i=0;i<n;i++)
 for(j=0;j<n;j++) 
 { c[i][j] = 0.0; 
   cref[i][j] = 0.0; 
   a[i][j] = drand48();
   b[i][j] = drand48();
 }
}

void compare(int N, double *wref, double *w)
{
double maxdiff,this_diff;
int numdiffs;
int i,j;
  numdiffs = 0;
  maxdiff = 0;
  for (i=0;i<N;i++)
   for (j=0;j<N;j++)
    {
     this_diff = wref[i*N+j]-w[i*N+j];
     if (this_diff < 0) this_diff = -1.0*this_diff;
     if (this_diff>threshold)
      { numdiffs++;
        if (this_diff > maxdiff) maxdiff=this_diff;
      }
    }
   if (numdiffs > 0)
      printf("%d Diffs found over threshold %f; Max Diff = %f\n",
               numdiffs,threshold,maxdiff);
   else
      printf("No differences found between reference and test versions\n");
}

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}
