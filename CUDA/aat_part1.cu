// Template for Programming Assignment 2
// Use "module load cuda" to enable compilation with the Nvidia C compiler nvcc
// Use "nvcc -O3" to compile code; this can be done even on OSC login node (does not have a GPU)
// To execute compiled code, you must either use a batch submission to run on a node with GPU
// or obtain an interactive GPU-node by using: qsub -I -l walltime=0:59:00 -l nodes=1:gpus=1


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#define threshold 1e-8
#define n (4096)
// Change n to 4096 for final testing; 
//#define n (1024)
// n is set to 256 since execution time of single thread template version is excessive
#define TILE_WIDTH 8

void init(void);
void ref(void);
void compare(int N, double *wref, double *w);
__global__ void test_kernel(int N, double *A, double *B, double *C);
double rtclock(void);

double a[n][n],b[n][n],c[n][n],cref[n][n];

int main(){

double clkbegin, clkend, t;
double *Ad,*Bd,*Cd;
int size;

  printf("Matrix Size = %d\n",n);

  init();
  clkbegin = rtclock();
  ref();
  clkend = rtclock();
  t = clkend-clkbegin;
  printf("Seq: Approx GFLOPS: %.1f ; Time = %.3f sec; cref[n/2][n/2-1] = %f; \n",
2.0*n*n*n/t/1e9,t,cref[n/2][n/2-1]);

  
  size = sizeof(double)*n*n;
  hipMalloc((void **) &Ad,size);
  hipMalloc((void **) &Bd,size);
  hipMalloc((void **) &Cd,size);
  hipMemcpy(Ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(Bd,b,size,hipMemcpyHostToDevice);
  clkbegin = rtclock();
  
  dim3 dimGrid(n/TILE_WIDTH, n/TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  
  test_kernel<<<dimGrid,dimBlock>>>(n,Ad,Bd,Cd);
  if (hipDeviceSynchronize() != hipSuccess) 
    printf ("Error return for test_kernel: Was execution done on a node with a GPU?\n");
  else
  {
   clkend = rtclock();
   t = clkend-clkbegin;
   hipMemcpy(c,Cd,size,hipMemcpyDeviceToHost);
   hipFree(Ad); hipFree(Bd); hipFree(Cd);
    printf("GPU: Approx GFLOPS: %.1f ; Time = %.3f sec; c[n/2][n/2-1] = %f; \n",
     2.0*n*n*n/t/1e9,t,c[n/2][n/2-1]);
    printf("Correctness Check for GPU solution:\n");
    compare(n, (double *) c,(double *) cref);
  }
}

__global__ void test_kernel(int N, double *A, double *B, double *C)
{

// Block Index along x & y
int bx = blockIdx.x; int by = blockIdx.y;
//Thread Index along x & y
int tx = threadIdx.x; int ty = threadIdx.y;

// Row & Column in resultant matrix C computed by the thread in the block
int Row = by * TILE_WIDTH + ty;
int Column = bx * TILE_WIDTH + tx;

double Pvalue = 0;

// Accumulate dot product
for (int k=0; k<N; ++k)
    Pvalue += A[Row*N+k]*B[Column*N+k];
    
// write final value to global memory
C[Row*N+Column] = Pvalue;

}

void ref(void)
{
int i,j,k;

  for (i=0;i<n;i++)
   for (j=0;j<n;j++)
    for(k=0;k<n;k++)
      cref[i][j] += a[i][k]*b[j][k];
}

void init(void)
{
int i,j;
for(i=0;i<n;i++)
 for(j=0;j<n;j++) 
 { c[i][j] = 0.0; 
   cref[i][j] = 0.0; 
   a[i][j] = drand48();
   b[i][j] = drand48();
 }
}

void compare(int N, double *wref, double *w)
{
double maxdiff,this_diff;
int numdiffs;
int i,j;
  numdiffs = 0;
  maxdiff = 0;
  for (i=0;i<N;i++)
   for (j=0;j<N;j++)
    {
     this_diff = wref[i*N+j]-w[i*N+j];
     if (this_diff < 0) this_diff = -1.0*this_diff;
     if (this_diff>threshold)
      { numdiffs++;
        if (this_diff > maxdiff) maxdiff=this_diff;
      }
    }
   if (numdiffs > 0)
      printf("%d Diffs found over threshold %f; Max Diff = %f\n",
               numdiffs,threshold,maxdiff);
   else
      printf("No differences found between reference and test versions\n");
}

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}
